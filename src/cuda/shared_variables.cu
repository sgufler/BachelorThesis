#include "hip/hip_runtime.h"


#include "thrust/device_vector.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "util.cuh"

thrust::device_vector<float> Xstar;	// guess vectors
thrust::device_vector<float> W;     // vector of inverse masses
thrust::device_vector<int> phase;


// textures
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;


extern "C"
{

	void freeSharedVectors()
	{
        Xstar.clear();
        W.clear();
        phase.clear();

        Xstar.shrink_to_fit();
        W.shrink_to_fit();
        phase.shrink_to_fit();
	}

    void appendPhaseAndMass(int *fase, float *w, uint numParticles)
    {
        int sizeW = W.size();

        // resize the vectors
        phase.resize(sizeW + numParticles);
        W.resize(sizeW + numParticles);

        // get raw pointers to the data
        int *dPhase = thrust::raw_pointer_cast(phase.data());
        float *dW = thrust::raw_pointer_cast(W.data());

        // copy the new data to the gpu
        copyArrayToDevice(dPhase + sizeW, fase, 0, numParticles * sizeof(int));
        copyArrayToDevice(dW + sizeW, w, 0, numParticles * sizeof(float));

        // resize but don't neet to fill
        Xstar.resize(4 * W.size());
    }

	void copyToXstar(float *pos, uint numParticles)
	{
        // copy X to X*
        float *dXstar = thrust::raw_pointer_cast(Xstar.data());
        checkCudaErrors(hipMemcpy((void*)dXstar, (void*)pos, numParticles*4*sizeof(float), hipMemcpyDeviceToDevice));
    }

    int *getPhaseRawPtr()
    {
        return thrust::raw_pointer_cast(phase.data());
    }

    float *getXstarRawPtr()
    {
        return thrust::raw_pointer_cast(Xstar.data());
    }

    float *getWRawPtr()
    {
        return thrust::raw_pointer_cast(W.data());
    }

    void printXstar()
    {
        printf("Xstar: size: %u\n", (uint)Xstar.size());
        thrust::device_ptr<float> d_Xstar(Xstar.data());
        uint index;
        for (uint i = 0; i < Xstar.size(); i++)
        {
            index = i * 4;
            printf("i: %u: %.2f, %.2f, %.2f\n", i, (float)*(d_Xstar + index + 0), (float)*(d_Xstar + index + 1), (float)*(d_Xstar + index + 2));
        }
        printf("\n");
    }

}
