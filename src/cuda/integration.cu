#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hiprand.h>
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>

#include "hip/hip_runtime_api.h"
#include "integration_kernel.cuh"
#include "util.cuh"
#include "shared_variables.cuh"

//#define PRINT

hiprandGenerator_t gen(0);

thrust::device_vector<float> V; // particle velocities
thrust::device_vector<float> lambda;
thrust::device_vector<float> denom;

thrust::device_vector<float> ros;

thrust::device_vector<uint> neighbors;
thrust::device_vector<uint> numNeighbors;
thrust::device_vector<uint> neighborsSdf;
thrust::device_vector<uint> numNeighborsSdf;

thrust::device_vector<float> textureVec;

float *rands;

extern "C"
{
    /*****************************************************************************
     *                              INITIALIZATION
     *****************************************************************************/


    void initIntegration()
    {
        allocateArray((void **)&rands, 6 * sizeof(float));
        checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    }

    void appendIntegrationParticle(float *v, float *ro, uint numParticles)
    {
        int sizeV = V.size();
        int sizeRo = ros.size();

        // resize the vectors
        V.resize(sizeV + 4 * numParticles);
        ros.resize(sizeRo + numParticles);

        // get raw pointers to the data
        float *dV = thrust::raw_pointer_cast(V.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

        // copy the new data over to the gpu
        copyArrayToDevice(dV + sizeV, v, 0, 4 * numParticles * sizeof(float));
        copyArrayToDevice(dRos + sizeRo, ro, 0, numParticles * sizeof(float));

        // resize but don't need to fill
        lambda.resize(ros.size());
        numNeighbors.resize(ros.size());
        neighbors.resize(V.size() * MAX_FLUID_NEIGHBORS);
        textureVec.resize(V.size());
    }

    void freeIntegrationVectors()
    {
         V.clear();
         lambda.clear();
         denom.clear();
         ros.clear();
         neighbors.clear();
         numNeighbors.clear();
         textureVec.clear();
         neighborsSdf.clear();
         numNeighborsSdf.clear();

         V.shrink_to_fit();
         lambda.shrink_to_fit();
         denom.shrink_to_fit();
         ros.shrink_to_fit();
         neighbors.shrink_to_fit();
         numNeighbors.shrink_to_fit();
         textureVec.shrink_to_fit();
         neighborsSdf.shrink_to_fit();
         numNeighborsSdf.shrink_to_fit();

         checkCudaErrors(hiprandDestroyGenerator(gen));
         freeArray(rands);
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, 1 * sizeof(SimParams)));
    }

















    /*****************************************************************************
     *                              UPDATE POSITIONS
     *****************************************************************************/

    void integrateSystem(float *pos, float deltaTime, uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_vel4((float4 *)thrust::raw_pointer_cast(V.data()));

        // copy current positions for reference later
        copyToXstar(pos, numParticles);

        // guess new positions based on forces
        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
            integrate_functor(deltaTime));
    }





















    /*****************************************************************************
     *                              BUILD GRID
     *****************************************************************************/

    void calcHash(uint *gridParticleHash, uint *gridParticleIndex, float *pos, int numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash, gridParticleIndex, (float4 *) pos, numParticles);

#ifdef PRINT
        printf("HASHES:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("particle: %u: hash: %u\n", i, (uint)*(dGPH + i));
        }
        printf("\n");
#endif

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedW,
                                     int   *sortedPhase,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        float *dW = getWRawPtr();
        int *dPhase = getPhaseRawPtr();

        float *pos;
        checkCudaErrors(hipMalloc((void**)&pos, numParticles*4*sizeof(float)));
        checkCudaErrors(hipMemcpy(pos, oldPos, numParticles*4*sizeof(float), hipMemcpyDeviceToDevice));

        if (sortedW != NULL)
        {
            checkCudaErrors(hipBindTexture(0, oldPosTex, pos, numParticles * sizeof(float4)));
            checkCudaErrors(hipBindTexture(0, invMassTex, dW, numParticles * sizeof(float)));
            checkCudaErrors(hipBindTexture(0, oldPhaseTex, dPhase, numParticles * sizeof(int)));
        }
        else
        {
            checkCudaErrors(hipBindTexture(0, posSdfTex, pos, numParticles * sizeof(float4)));
        }

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cellStart,
                                                                           cellEnd,
                                                                           (float4 *) sortedPos,
                                                                           sortedW,
                                                                           sortedPhase,
                                                                           gridParticleHash,
                                                                           gridParticleIndex,
                                                                           (float4 *) oldPos,
                                                                           dW,
                                                                           dPhase,
                                                                           numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");
        
#ifdef PRINT
        /*printf("Sorted:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        thrust::device_ptr<uint> dGPI(gridParticleIndex);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("i: %u: hash: %u\n", i, (uint)*(dGPH + i));
            printf("i: %u: part: %u\n", i, (uint)*(dGPI + i));
        }
        printf("\n");*/


        printf("Sorted:\n");
        thrust::device_ptr<uint> dstart(cellStart);
        thrust::device_ptr<uint> dend(cellEnd);
        for (uint i = 0; i < 32; i++)
        {
            printf("i: %u: start: %u\n", i, (uint)*(dstart + i));
            printf("i: %u: end: %u\n", i, (uint)*(dend + i));
        }
        printf("\n");
#endif
        
        if (sortedW != NULL)
        {
            checkCudaErrors(hipUnbindTexture(oldPosTex));
            checkCudaErrors(hipUnbindTexture(invMassTex));
            checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        }
        else
        {
            checkCudaErrors(hipUnbindTexture(posSdfTex));
        }

        checkCudaErrors(hipFree(pos));
    }

    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }

































    /*****************************************************************************
     *                              PROCESS COLLISIONS
     *****************************************************************************/

    void sortByType(float *dPos, uint numParticles)
    {

    }

    void collideWorld(float *pos, float *sortedPos, uint numParticles, int3 minBounds, int3 maxBounds)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_Xstar((float4*)getXstarRawPtr());
        thrust::device_ptr<int> d_phase(getPhaseRawPtr());

        // create random vars for boundary collisions
        checkCudaErrors(hiprandGenerateUniform(gen, rands, 6));

        // check for boundary collisions and move particles
//        thrust::for_each
//        thrust::transform(d_pos4, d_pos4 + numParticles, d_Xstar, d_pos4, collide_world_functor(rands, minBounds, maxBounds));

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_Xstar, d_phase)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_Xstar+numParticles, d_phase+numParticles)),
            collide_world_functor(rands, minBounds, maxBounds));
    }

    void collide(float *particles,
                 float *sortedPos,
                 float *sortedW,
                 int   *sortedPhase,
                 float *sortedPosSdf,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint  *cellStartSdf,
                 uint  *cellEndSdf,
                 uint   numParticles,
                 uint   numParticlesSdf,
                 uint   numCells)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(int)));
        checkCudaErrors(hipBindTexture(0, posSdfTex, sortedPosSdf, numParticlesSdf * sizeof(float4)));

        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellStartSdfTex, cellStartSdf, numCells * sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndSdfTex, cellEndSdf, numCells * sizeof(uint)));
        
        // store neighbors
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dXstar = getXstarRawPtr();

        numNeighborsSdf.resize(numParticles);
        neighborsSdf.resize(numParticles * MAX_SDF_NEIGHBORS);
        uint *dNeighborsSdf = thrust::raw_pointer_cast(neighborsSdf.data());
        uint *dNumNeighborsSdf = thrust::raw_pointer_cast(numNeighborsSdf.data());

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)particles,
                                              (float4 *)dXstar,
                                              gridParticleIndex,
                                              numParticles,
                                              numParticlesSdf,
                                              dNeighbors,
                                              dNumNeighbors,
                                              dNeighborsSdf,
                                              dNumNeighborsSdf);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        checkCudaErrors(hipUnbindTexture(posSdfTex));

        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
        checkCudaErrors(hipUnbindTexture(cellStartSdfTex));
        checkCudaErrors(hipUnbindTexture(cellEndSdfTex));
    }

























    /*****************************************************************************
     *                              UPDATE VELOCITIES
     *****************************************************************************/

    void calcVelocity(float *dpos, float deltaTime, uint numParticles)
    {
        float *dXstar = getXstarRawPtr();
        thrust::device_ptr<float4> d_Xstar((float4*)dXstar);
        thrust::device_ptr<float4> d_pos((float4*)dpos);
        thrust::device_ptr<float4> d_vel((float4*)thrust::raw_pointer_cast(V.data()));


        thrust::transform(d_pos, d_pos + numParticles, d_Xstar, d_vel, subtract_functor(deltaTime));

    }























    /*****************************************************************************
     *                              SOLVE FLUIDS
     *****************************************************************************/
    void solveFluids(float *sortedPos,
                     float *sortedW,
                     int   *sortedPhase,
                     uint  *gridParticleIndex,
                     uint  *cellStart,
                     uint  *cellEnd,
                     float *particles,
                     uint   numParticles,
                     uint   numCells)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        float *dLambda = thrust::raw_pointer_cast(lambda.data());
//        float *dDenom = thrust::raw_pointer_cast(denom.data());
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

//        printf("ros: %u, numParts: %u\n", (uint)ros.size(), numParticles);

        // execute the kernel
        findLambdasD<<< numBlocks, numThreads >>>(dLambda,
                                                  gridParticleIndex,
                                                  cellStart,
                                                  cellEnd,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // execute the kernel
        solveFluidsD<<< numBlocks, numThreads >>>(dLambda,
                                                  gridParticleIndex,
                                                  (float4 *) particles,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }
}
